#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <stdio.h>
#include <time.h>


// Thread block size
#define BLOCK_SIZE 16

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
typedef struct {
    int width;
    int height;
    int stride;
    float* elements;
} Matrix;

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col,
    float value)
{
    A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
__device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width = BLOCK_SIZE;
    Asub.height = BLOCK_SIZE;
    Asub.stride = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
        + BLOCK_SIZE * col];
    return Asub;
}




// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);


void MatMul(const Matrix A, const Matrix B, Matrix C);

int main()
{
    int size = 16;

    Matrix A;
    Matrix B;
    Matrix C;

    A.width = size;
    B.width = size;
    C.width = size;

    A.height = size;
    B.height = size;
    C.height = size;

    A.elements = (float*)malloc(sizeof(float) * size * size);
    B.elements = (float*)malloc(sizeof(float) * size * size);
    C.elements = (float*)malloc(sizeof(float) * size * size);

    srand(time(NULL));

    for (int i = 0; i < size * size; i++) {
        A.elements[i] =  1;
        B.elements[i] =  1;
        C.elements[i] = 0;
    }

    MatMul(A, B, C);

    for (int i = 0; i < C.width; i++) {
        for (int j = 0; j < C.height; j++) {
            printf("%f ", C.elements[j * C.width + i]);
        }
        printf("\n");
    }

    return 0;
}

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width;
    d_A.stride = A.width;
    d_A.height = A.height;

    size_t size = A.width * A.height * sizeof(float);

    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

    Matrix d_B;
    d_B.width = B.width;
    d_B.stride = B.width;
    d_B.height = B.height;

    size = B.width * B.height * sizeof(float);

    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width;
    d_C.stride = C.width;
    d_C.height = C.height;

    size = C.width * C.height * sizeof(float);

    hipMalloc(&d_C.elements, size);

    // Invoke kernel

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel << <dimGrid, dimBlock >> > (d_A, d_B, d_C);

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);


    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (A.width / BLOCK_SIZE); m++) {

        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);

        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        // Use shared memory to store Asub and Bsub respectively
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();

        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCK_SIZE; e++)
            Cvalue += As[row][e] * Bs[e][col];


        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }
    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}



